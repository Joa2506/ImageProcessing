#include "Engine.hpp"
#include <ImageProcessing.cuh>
#include "utils.hpp"
bool Engine::convertToGray()
{

    const int colorBytes = mImageInput.step * mImageInput.rows;
	const int grayBytes = mImageOutput.step * mImageOutput.rows;

    unsigned char * d_input, *d_output;

    gpuErrchk(hipMalloc<unsigned char>(&d_input, colorBytes));
    gpuErrchk(hipMalloc<unsigned char>(&d_output, grayBytes));
    gpuErrchk(hipMemcpy(d_input, mImageInput.ptr(), colorBytes, hipMemcpyHostToDevice));
    printf("Images copied to GPU\n");

    const dim3 block(16,16);
    const dim3 grid((mImageInput.cols + block.x -1)/block.x, (mImageInput.rows + block.y - 1)/block.y);

    bgrToGray <<<grid, block>>>(d_input, d_output, mImageInput.cols, mImageInput.rows, mImageInput.step, mImageOutput.step);
    
    //hipDeviceSynchronize();
    gpuErrchk(hipDeviceSynchronize());
    
    gpuErrchk(hipMemcpy(mImageOutput.ptr(), d_output, grayBytes, hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(d_input));
    gpuErrchk(hipFree(d_output));

    fflush(stdout);

    return true;
}


bool Engine::gaussianBlur()
{

    const int colorBytes = mImageInput.step * mImageInput.rows;
	
    unsigned char * d_input, *d_output;

    gpuErrchk(hipMalloc<unsigned char>(&d_input, colorBytes));
    gpuErrchk(hipMalloc<unsigned char>(&d_output, colorBytes));
    gpuErrchk(hipMemcpy(d_input, mImageInput.ptr(), colorBytes, hipMemcpyHostToDevice));
    printf("Images copied to GPU\n");

    const dim3 block(16,16);
    const dim3 grid((mImageInput.cols + block.x -1)/block.x, (mImageInput.rows + block.y - 1)/block.y);

    gaussianBlurKernel <<<grid, block>>>(d_input, d_output, mImageInput.cols, mImageInput.rows);
    
    //hipDeviceSynchronize();
    gpuErrchk(hipDeviceSynchronize());
    
    gpuErrchk(hipMemcpy(mImageOutput.ptr(), d_output, colorBytes, hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(d_input));
    gpuErrchk(hipFree(d_output));

    fflush(stdout);

    return true;
}

void Engine::brightness(int brightnessLevel)
{
    //Size of image in bytes
    const int bytes = mImageInput.cols * mImageInput.rows * sizeof(unsigned char);

    //Device buffers
    unsigned char * d_input, *d_output;

    gpuErrchk(hipMalloc<unsigned char>(&d_input, bytes));
    gpuErrchk(hipMalloc<unsigned char>(&d_output, bytes));

    gpuErrchk(hipMemcpy(d_input, mImageInput.ptr(), bytes, hipMemcpyHostToDevice));
    printf("Image uploaded to GPU\n");

    const dim3 block(16,16);
    const dim3 grid((mImageInput.cols + block.x -1)/block.x, (mImageInput.rows + block.y - 1)/block.y);

    brightnessKernel <<<grid, block>>>(d_input, d_output, mImageInput.cols, mImageInput.rows, brightnessLevel, mImageInput.step);

    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(mImageOutput.ptr(), d_output, bytes, hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(d_input));
    gpuErrchk(hipFree(d_output));


}
